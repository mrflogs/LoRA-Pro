#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// SPDX-License-Identifier: Apache-2.0

// DeepSpeed Team

#include <stdexcept>
#include "context.h"
#include "fp_quantize.h"
#include "memory_access_utils.h"
#include "reduction_utils.h"

#include <hip/hip_runtime.h>
#include <stdint.h>

#include <hip/hip_fp16.h>
#include <hiprand/hiprand_kernel.h>

#include <hip/hip_bf16.h>
#include <hip/hip_runtime_api.h>

using ROp = reduce::ROpType;

namespace quantization {

constexpr int access_granularity = 16;
constexpr int quanitzed_access_granularity = 4;
constexpr int quanitzed_access_granularity_6bits = 2;
constexpr int threads = 256;
constexpr int warps = threads / 32;

}  // namespace quantization

template <int _mantisa_bits, int q_mantisa_bits, int stochastic_rounding>
__device__ void round(uint32_t& mantisa, uint32_t& dst_exponent, hiprandStatePhilox4_32_10_t* state)
{
    constexpr uint32_t mantisa_mask = (1 << (_mantisa_bits - q_mantisa_bits)) - 1;
    uint32_t offset = stochastic_rounding ? (hiprand_poisson(state, 10) & mantisa_mask)
                                          : 1 << (_mantisa_bits - q_mantisa_bits - 1);
    mantisa += offset;
    dst_exponent += (((mantisa & ~mantisa_mask) == (1 << _mantisa_bits)) ? 1 : 0);
}

template <int _mantisa_bits, int _exponent_bits, int q_mantisa_bits, int q_exponent_bits>
__device__ void clip(uint32_t& exponent, uint32_t& mantisa)
{
    constexpr uint32_t max_exponent = (1 << (q_exponent_bits - 1)) + (1 << (_exponent_bits - 1));
    constexpr uint32_t min_exponent =
        (1 << (_exponent_bits - 1)) - ((1 << (q_exponent_bits - 1)) - 1);
    if (exponent > max_exponent) {
        exponent = max_exponent;
        mantisa = (((uint32_t)-1) >> (32 - q_mantisa_bits)) << 1;  //.11 .. 10
    }
    if (exponent < min_exponent) {
        exponent = min_exponent;
        mantisa = 0;
    }
}

template <typename T,
          int unroll,
          int _mantisa_bits,
          int _exponent_bits,
          int total_q_bits = 8,
          int q_mantisa_bits = 3,
          int stochastic_rounding = 0>
__global__ void apply_quantization(T* val,
                                   uint8_t* q_val,
                                   int group_size,
                                   std::pair<uint64_t, uint64_t> seed,
                                   float q_range)
{
    int tidx = threadIdx.x;
    int wid = tidx >> 5;
    int lane = tidx & 0x1f;
    int gid = blockIdx.x * quantization::warps + wid;

    constexpr int q_exponent_bits = total_q_bits - q_mantisa_bits - 1;
    constexpr uint32_t _mantisa_mask = (1 << _mantisa_bits) - 1;
    constexpr uint32_t _exponent_mask = ((1 << _exponent_bits) - 1) << _mantisa_bits;
    constexpr uint32_t _sign_mask = 1 << (_mantisa_bits + _exponent_bits);
    // CG helpers
    cg::thread_block tb = cg::this_thread_block();
    cg::thread_block_tile<hw_warp_size> warp = cg::tiled_partition<hw_warp_size>(tb);

    constexpr uint32_t vector_size = quantization::access_granularity / sizeof(T);
    constexpr uint32_t load_stride = vector_size * hw_warp_size;
    constexpr uint32_t store_stride = (total_q_bits * vector_size / 8) * hw_warp_size;
    const uint32_t thread_offset = lane * vector_size;
    const uint32_t store_thread_offset = lane * (total_q_bits * vector_size / 8);
    const uint32_t base_load_offset = gid * group_size + thread_offset;
    const uint32_t base_store_offset =
        gid * ((group_size * total_q_bits / 8) + 4) +
        store_thread_offset;  // 4-byte for saving the scale per group
    const T* load_base_ptr = val + base_load_offset;
    T tmp_buf[unroll * vector_size];
    T cur_max;
    reduce::init<ROp::Max>(&cur_max);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed.first, idx, seed.second, &state);

#pragma unroll
    for (int i = 0; i < unroll; i++) {
        if (i * load_stride + thread_offset < group_size) {
            mem_access::load_global<quantization::access_granularity>(
                &tmp_buf[vector_size * i], load_base_ptr + i * load_stride);
            for (int j = 0; j < vector_size; j++)
                cur_max = reduce::element<ROp::Max>(cur_max, __habs(tmp_buf[i * vector_size + j]));
        }
    }
    reduce::_block<T, 1, ROp::Max>(tb, warp, &cur_max);

    int mantisa_mask = ((1 << q_mantisa_bits) - 1);
    mantisa_mask <<= (_mantisa_bits - q_mantisa_bits);

    uint8_t* store_base_ptr = q_val + base_store_offset;
    float scale = (float)q_range / conversion::to<float>(cur_max);
#pragma unroll
    for (int i = 0; i < unroll; i++) {
        if (i * load_stride + thread_offset < group_size) {
            uint64_t q_buf = 0;
            uint64_t q_buf1 = 0;
#pragma unroll
            for (int j = 0; j < vector_size; j++) {
                float val_f = conversion::to<float>(tmp_buf[i * vector_size + j]) * scale;
                uint32_t* data = reinterpret_cast<uint32_t*>(&val_f);
                uint32_t sign = (data[0] & _sign_mask) >> (_mantisa_bits + _exponent_bits);
                uint32_t cur_exponent = (data[0] & _exponent_mask) >> _mantisa_bits;
                uint32_t dst_mantisa = (data[0] & _mantisa_mask);

                uint32_t dst_exponent = cur_exponent;

                round<_mantisa_bits, q_mantisa_bits, stochastic_rounding>(
                    dst_mantisa, dst_exponent, &state);
                if (cur_exponent != 0)
                    clip<_mantisa_bits, _exponent_bits, q_mantisa_bits, q_exponent_bits>(
                        dst_exponent, dst_mantisa);

                dst_mantisa = (dst_mantisa & mantisa_mask) >> (_mantisa_bits - q_mantisa_bits);

                if (dst_exponent != (1 << q_exponent_bits) - 1)
                    dst_exponent = (dst_exponent - ((1 << (_exponent_bits - 1)) - 1)) +
                                   (1 << (q_exponent_bits - 1)) - 1;
                if (total_q_bits == 8 || total_q_bits == 4 || total_q_bits == 6)
                    q_buf = q_buf |
                            ((uint64_t)((uint8_t)(sign << (q_exponent_bits + q_mantisa_bits) |
                                                  (dst_exponent << q_mantisa_bits) | dst_mantisa))
                             << j * total_q_bits);
                else if (total_q_bits == 12) {
                    if (j < 5)
                        q_buf =
                            q_buf |
                            ((uint64_t)((uint16_t)(sign << (q_exponent_bits + q_mantisa_bits) |
                                                   (dst_exponent << q_mantisa_bits) | dst_mantisa))
                             << j * total_q_bits);
                    else
                        q_buf1 =
                            q_buf1 |
                            ((uint64_t)((uint16_t)(sign << (q_exponent_bits + q_mantisa_bits) |
                                                   (dst_exponent << q_mantisa_bits) | dst_mantisa))
                             << (j - 5) * total_q_bits);
                }
            }
            if (total_q_bits == 12) {
                uint64_t last_nibble_mask = 0xf;
                last_nibble_mask = q_buf1 & last_nibble_mask;
                q_buf = (last_nibble_mask << 60) | q_buf;
                q_buf1 >>= 4;
            }
            uint8_t* int8_data = reinterpret_cast<uint8_t*>(&q_buf);
            uint8_t* int8_data1 = reinterpret_cast<uint8_t*>(&q_buf1);
            if (total_q_bits == 6) {
                mem_access::store_global<quantization::quanitzed_access_granularity_6bits>(
                    store_base_ptr + i * store_stride, int8_data);
                mem_access::store_global<quantization::quanitzed_access_granularity_6bits>(
                    store_base_ptr + i * store_stride +
                        quantization::quanitzed_access_granularity_6bits,
                    int8_data + quantization::quanitzed_access_granularity_6bits);
                mem_access::store_global<quantization::quanitzed_access_granularity_6bits>(
                    store_base_ptr + i * store_stride +
                        quantization::quanitzed_access_granularity_6bits * 2,
                    int8_data + 2 * quantization::quanitzed_access_granularity_6bits);
            } else {
                mem_access::store_global<quantization::quanitzed_access_granularity>(
                    store_base_ptr + i * store_stride, int8_data);

                if (total_q_bits > 4) {
                    mem_access::store_global<quantization::quanitzed_access_granularity>(
                        store_base_ptr + i * store_stride +
                            quantization::quanitzed_access_granularity,
                        int8_data + quantization::quanitzed_access_granularity);
                    if (total_q_bits == 12) {
                        mem_access::store_global<quantization::quanitzed_access_granularity>(
                            store_base_ptr + i * store_stride +
                                quantization::quanitzed_access_granularity * 2,
                            int8_data1);
                    }
                }
            }
        }
    }
    if (lane == 0) {
        float q_scale = conversion::to<float>(cur_max) / (float)q_range;
        uint8_t* scale_as_int8 = reinterpret_cast<uint8_t*>(&q_scale);
        uint32_t scale_offset =
            gid * ((group_size * total_q_bits / 8) + 4) + (group_size * total_q_bits / 8);
        if (total_q_bits != 6)
            mem_access::store_global<quantization::quanitzed_access_granularity>(
                q_val + scale_offset, scale_as_int8);
        else {
            mem_access::store_global<quantization::quanitzed_access_granularity_6bits>(
                q_val + scale_offset, scale_as_int8);
            mem_access::store_global<quantization::quanitzed_access_granularity_6bits>(
                q_val + scale_offset + quantization::quanitzed_access_granularity_6bits,
                scale_as_int8 + quantization::quanitzed_access_granularity_6bits);
        }
    }
}

template <typename T,
          int q_mantisa_bits,
          int total_q_bits = 16,
          int _mantisa_bits = 3,
          int _exponent_bits = 4>
__global__ void apply_dequantization(uint8_t* val, T* q_val, int group_size, int total_num_elements)
{
    constexpr uint32_t vector_size = quantization::access_granularity / sizeof(T);
    int tidx = (blockIdx.x * blockDim.x + threadIdx.x) * vector_size;

    constexpr int quantized_bits = _mantisa_bits + _exponent_bits + 1;
    constexpr int q_exponent_bits = total_q_bits - q_mantisa_bits - 1;
    constexpr uint16_t _mantisa_mask = (1 << _mantisa_bits) - 1;
    constexpr uint16_t _exponent_mask = ((1 << _exponent_bits) - 1) << _mantisa_bits;
    constexpr uint16_t _sign_mask = 1 << (_mantisa_bits + _exponent_bits);
    const uint32_t g_index = (tidx / group_size);
    const uint32_t group_size_bytes = (group_size * quantized_bits / 8);
    const uint8_t* load_base_ptr =
        val + g_index * (group_size_bytes + 4) + (tidx % group_size) * quantized_bits / 8;

    int mantisa_mask = ((1 << q_mantisa_bits) - 1);
    mantisa_mask <<= (_mantisa_bits - q_mantisa_bits);

    T* store_base_ptr = q_val + tidx;
    float scale;

    uint8_t* scale_as_int8 = reinterpret_cast<uint8_t*>(&scale);
    if (quantized_bits == 6) {
        mem_access::load_global<quantization::quanitzed_access_granularity>(
            scale_as_int8, val + g_index * (group_size_bytes + 4) + group_size_bytes);
        mem_access::load_global<quantization::quanitzed_access_granularity_6bits>(
            scale_as_int8 + quantization::quanitzed_access_granularity_6bits,
            val + g_index * (group_size_bytes + 4) + group_size_bytes +
                quantization::quanitzed_access_granularity_6bits);
    } else
        mem_access::load_global<quantization::quanitzed_access_granularity>(
            scale_as_int8, val + g_index * (group_size_bytes + 4) + group_size_bytes);

    if (tidx < total_num_elements) {
        uint64_t q_buf_in;
        uint64_t q_buf_in1;
        uint8_t* int8_data = reinterpret_cast<uint8_t*>(&q_buf_in);
        uint8_t* int8_data1 = reinterpret_cast<uint8_t*>(&q_buf_in1);
        if (quantized_bits == 6) {
            mem_access::load_global<quantization::quanitzed_access_granularity_6bits>(
                int8_data, load_base_ptr);
            mem_access::load_global<quantization::quanitzed_access_granularity_6bits>(
                int8_data + quantization::quanitzed_access_granularity_6bits,
                load_base_ptr + quantization::quanitzed_access_granularity_6bits);
            mem_access::load_global<quantization::quanitzed_access_granularity_6bits>(
                int8_data + quantization::quanitzed_access_granularity_6bits * 2,
                load_base_ptr + quantization::quanitzed_access_granularity_6bits * 2);

        } else {
            mem_access::load_global<quantization::quanitzed_access_granularity>(int8_data,
                                                                                load_base_ptr);
            if (quantized_bits > 4) {
                mem_access::load_global<quantization::quanitzed_access_granularity>(
                    int8_data + quantization::quanitzed_access_granularity,
                    load_base_ptr + quantization::quanitzed_access_granularity);
                if (quantized_bits == 12) {
                    mem_access::load_global<quantization::quanitzed_access_granularity>(
                        int8_data1, load_base_ptr + quantization::quanitzed_access_granularity * 2);
                }
            }
        }
        T store_buf[vector_size];
        uint16_t* q_buf = reinterpret_cast<uint16_t*>(store_buf);
#pragma unroll
        for (int j = 0; j < vector_size; j++) {
            uint16_t new_data;
            if (j < 5 || quantized_bits != 12) {
                new_data = (uint16_t)(q_buf_in >> (j * quantized_bits));
            } else {
                if (j == 5) {
                    new_data = (uint16_t)(q_buf_in1);
                    new_data = (uint16_t)((new_data << 4) | (q_buf_in >> 60));
                } else
                    new_data = (uint16_t)(q_buf_in1 >> ((j - 6) * quantized_bits + 8));
            }

            uint16_t sign = (new_data & _sign_mask) >> (_mantisa_bits + _exponent_bits);
            uint16_t dst_exponent = (new_data & _exponent_mask) >> _mantisa_bits;
            uint16_t dst_mantisa = (new_data & _mantisa_mask);

            if (dst_exponent != (1 << q_exponent_bits) - 1)
                dst_exponent = (dst_exponent - ((1 << (_exponent_bits - 1)) - 1)) +
                               (1 << (q_exponent_bits - 1)) - 1;

            q_buf[j] =
                ((sign << (q_exponent_bits + q_mantisa_bits)) | (dst_exponent << q_mantisa_bits) |
                 (dst_mantisa << (q_mantisa_bits - _mantisa_bits)));
            float up_cast = conversion::to<float>(store_buf[j]);
            store_buf[j] = conversion::to<T>(up_cast * scale);
        }
        mem_access::store_global<quantization::access_granularity>(store_base_ptr, store_buf);
    }
}

#define LAUNCH_FOR_QUANTIZATION_UNROLL(COUNT)                                    \
    case COUNT:                                                                  \
        apply_quantization<T,                                                    \
                           COUNT,                                                \
                           mantisa,                                              \
                           exponent,                                             \
                           CONST_Q_BITS,                                         \
                           CONST_Q_MANTISA_BITS,                                 \
                           CONST_STOCHASTIC_ROUNDING>                            \
            <<<grid, block, 0, stream>>>(val, q_val, group_size, seed, q_range); \
        break;

template <typename T, int mantisa, int exponent>
void launch_quantization(T* val,
                         uint8_t* q_val,
                         int num_groups,
                         int group_size,
                         hipStream_t stream,
                         float q_range,
                         int q_bits,
                         int q_mantisa_bits,
                         int stochastic_rounding)
{
    const dim3 grid((num_groups + quantization::warps - 1) / quantization::warps);
    const dim3 block(quantization::threads);

    std::pair<uint64_t, uint64_t> seed = FPContext::Instance().IncrementOffset(16);

    constexpr int vals_per_unroll = hw_warp_size * quantization::access_granularity / sizeof(T);

    const int copy_unroll = (group_size + vals_per_unroll - 1) / vals_per_unroll;
    QUANT_SWITCH((q_bits - q_mantisa_bits - 1) * q_mantisa_bits + stochastic_rounding, [&] {
        switch (copy_unroll) {
            LAUNCH_FOR_QUANTIZATION_UNROLL(1)
            LAUNCH_FOR_QUANTIZATION_UNROLL(2)
            LAUNCH_FOR_QUANTIZATION_UNROLL(3)
            LAUNCH_FOR_QUANTIZATION_UNROLL(4)
            LAUNCH_FOR_QUANTIZATION_UNROLL(5)
            LAUNCH_FOR_QUANTIZATION_UNROLL(6)
        }
    });
}
#define INSTANTIATE_LAUNCH_QUANTIZATION(T, mantisa, exponent) \
    template void launch_quantization<T, mantisa, exponent>(  \
        T*, uint8_t*, int, int, hipStream_t, float q_range, int, int, int);
// fp8(E4M3), nearest-rounding
#ifdef BF16_AVAILABLE
INSTANTIATE_LAUNCH_QUANTIZATION(__hip_bfloat16, 23, 8);
#endif
INSTANTIATE_LAUNCH_QUANTIZATION(__half, 23, 8);

template <typename T, int mantisa>
void launch_dequantization(uint8_t* val,
                           T* q_val,
                           int num_groups,
                           int group_size,
                           int q_mantisa_bits,
                           int q_exponent_bits,
                           hipStream_t stream)
{
    int blocks = ((num_groups * group_size) - 1) /
                     (quantization::threads * (quantization::access_granularity / sizeof(T))) +
                 1;
    const dim3 grid(blocks);
    const dim3 block(quantization::threads);
    DEQUANT_SWITCH(q_mantisa_bits * q_exponent_bits, [&] {
        apply_dequantization<T, mantisa, 16, CONST_Q_MANTISA_BITS, CONST_Q_EXPONENT_BITS>
            <<<grid, block, 0, stream>>>(val, q_val, group_size, (num_groups * group_size));
    });
}
#define INSTANTIATE_LAUNCH_DEQUANTIZATION(T, mantisa) \
    template void launch_dequantization<T, mantisa>(uint8_t*, T*, int, int, int, int, hipStream_t);
// fp8(E4M3)
#ifdef BF16_AVAILABLE
INSTANTIATE_LAUNCH_DEQUANTIZATION(__hip_bfloat16, 7);
#endif
INSTANTIATE_LAUNCH_DEQUANTIZATION(__half, 10);

template <typename T,
          int q_mantisa_bits,
          int total_q_bits = 16,
          int _mantisa_bits = 3,
          int _exponent_bits = 4>
__global__ void apply_selective_dequantization(uint8_t* val,
                                               T* q_val,
                                               int32_t* indexes,
                                               int group_size,
                                               int total_num_elements)
{
    int index = indexes[blockIdx.x];
    constexpr uint32_t vector_size = quantization::access_granularity / sizeof(T);
    int tidx = (blockIdx.y * blockDim.x + threadIdx.x) * vector_size;
    int input_index = index * total_num_elements + tidx;
    constexpr int quantized_bits = _mantisa_bits + _exponent_bits + 1;
    constexpr int q_exponent_bits = total_q_bits - q_mantisa_bits - 1;
    constexpr uint16_t _mantisa_mask = (1 << _mantisa_bits) - 1;
    constexpr uint16_t _exponent_mask = ((1 << _exponent_bits) - 1) << _mantisa_bits;
    constexpr uint16_t _sign_mask = 1 << (_mantisa_bits + _exponent_bits);
    const uint32_t g_index = (input_index / group_size);
    const uint32_t group_size_bytes = (group_size * quantized_bits / 8);
    const uint8_t* load_base_ptr =
        val + g_index * (group_size_bytes + 4) + (input_index % group_size) * quantized_bits / 8;

    int mantisa_mask = ((1 << q_mantisa_bits) - 1);
    mantisa_mask <<= (_mantisa_bits - q_mantisa_bits);

    T* store_base_ptr = q_val + tidx + blockIdx.x * total_num_elements;
    float scale;

    uint8_t* scale_as_int8 = reinterpret_cast<uint8_t*>(&scale);
    if (quantized_bits == 6) {
        mem_access::load_global<quantization::quanitzed_access_granularity>(
            scale_as_int8, val + g_index * (group_size_bytes + 4) + group_size_bytes);
        mem_access::load_global<quantization::quanitzed_access_granularity_6bits>(
            scale_as_int8 + quantization::quanitzed_access_granularity_6bits,
            val + g_index * (group_size_bytes + 4) + group_size_bytes +
                quantization::quanitzed_access_granularity_6bits);
    } else
        mem_access::load_global<quantization::quanitzed_access_granularity>(
            scale_as_int8, val + g_index * (group_size_bytes + 4) + group_size_bytes);

    if (tidx < total_num_elements) {
        uint64_t q_buf_in;
        uint64_t q_buf_in1;
        uint8_t* int8_data = reinterpret_cast<uint8_t*>(&q_buf_in);
        uint8_t* int8_data1 = reinterpret_cast<uint8_t*>(&q_buf_in1);
        if (quantized_bits == 6) {
            mem_access::load_global<quantization::quanitzed_access_granularity_6bits>(
                int8_data, load_base_ptr);
            mem_access::load_global<quantization::quanitzed_access_granularity_6bits>(
                int8_data + quantization::quanitzed_access_granularity_6bits,
                load_base_ptr + quantization::quanitzed_access_granularity_6bits);
            mem_access::load_global<quantization::quanitzed_access_granularity_6bits>(
                int8_data + quantization::quanitzed_access_granularity_6bits * 2,
                load_base_ptr + quantization::quanitzed_access_granularity_6bits * 2);
        } else {
            mem_access::load_global<quantization::quanitzed_access_granularity>(int8_data,
                                                                                load_base_ptr);
            if (quantized_bits > 4) {
                mem_access::load_global<quantization::quanitzed_access_granularity>(
                    int8_data + quantization::quanitzed_access_granularity,
                    load_base_ptr + quantization::quanitzed_access_granularity);
                if (quantized_bits == 12) {
                    mem_access::load_global<quantization::quanitzed_access_granularity>(
                        int8_data1, load_base_ptr + quantization::quanitzed_access_granularity * 2);
                }
            }
        }
        T store_buf[vector_size];
        uint16_t* q_buf = reinterpret_cast<uint16_t*>(store_buf);
#pragma unroll
        for (int j = 0; j < vector_size; j++) {
            uint16_t new_data;
            if (j < 5 || quantized_bits != 12) {
                new_data = (uint16_t)(q_buf_in >> (j * quantized_bits));
            } else {
                if (j == 5) {
                    new_data = (uint16_t)(q_buf_in1);
                    new_data = (uint16_t)((new_data << 4) | (q_buf_in >> 60));
                } else
                    new_data = (uint16_t)(q_buf_in1 >> ((j - 6) * quantized_bits + 8));
            }

            uint16_t sign = (new_data & _sign_mask) >> (_mantisa_bits + _exponent_bits);
            uint16_t dst_exponent = (new_data & _exponent_mask) >> _mantisa_bits;
            uint16_t dst_mantisa = (new_data & _mantisa_mask);

            if (dst_exponent != (1 << q_exponent_bits) - 1)
                dst_exponent = (dst_exponent - ((1 << (_exponent_bits - 1)) - 1)) +
                               (1 << (q_exponent_bits - 1)) - 1;

            q_buf[j] =
                ((sign << (q_exponent_bits + q_mantisa_bits)) | (dst_exponent << q_mantisa_bits) |
                 (dst_mantisa << (q_mantisa_bits - _mantisa_bits)));
            float up_cast = conversion::to<float>(store_buf[j]);
            store_buf[j] = conversion::to<T>(up_cast * scale);
        }
        mem_access::store_global<quantization::access_granularity>(store_base_ptr, store_buf);
    }
}

template <typename T, int mantisa>
void launch_selective_dequantization(uint8_t* val,
                                     T* q_val,
                                     int32_t* indexes,
                                     int num_groups,
                                     int group_size,
                                     int num_indexes,
                                     int q_mantisa_bits,
                                     int q_exponent_bits,
                                     hipStream_t stream)
{
    int total_elements_per_index = (num_groups / num_indexes) * group_size;
    int blocks = (total_elements_per_index - 1) /
                     (quantization::threads * (quantization::access_granularity / sizeof(T))) +
                 1;
    const dim3 grid(num_indexes, blocks);
    const dim3 block(quantization::threads);
    DEQUANT_SWITCH(q_mantisa_bits * q_exponent_bits, [&] {
        apply_selective_dequantization<T, mantisa, 16, CONST_Q_MANTISA_BITS, CONST_Q_EXPONENT_BITS>
            <<<grid, block, 0, stream>>>(val, q_val, indexes, group_size, total_elements_per_index);
    });
}
#define INSTANTIATE_LAUNCH_SELECTIVE_DEQUANTIZATION(T, mantisa) \
    template void launch_selective_dequantization<T, mantisa>(  \
        uint8_t*, T*, int32_t*, int, int, int, int, int, hipStream_t);
// fp8(E4M3)
#ifdef BF16_AVAILABLE
INSTANTIATE_LAUNCH_SELECTIVE_DEQUANTIZATION(__hip_bfloat16, 7);
#endif
INSTANTIATE_LAUNCH_SELECTIVE_DEQUANTIZATION(__half, 10);
