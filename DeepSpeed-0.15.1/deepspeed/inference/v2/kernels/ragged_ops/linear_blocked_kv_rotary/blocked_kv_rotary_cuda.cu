#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// SPDX-License-Identifier: Apache-2.0

// DeepSpeed Team

#include <cassert>
#include "blocked_kv_rotary.cuh"
#include "conversion_utils.h"
#include "ds_kernel_utils.h"
#include "memory_access_utils.h"

namespace cg = cooperative_groups;

namespace kv_rot {

constexpr int granularity = 16;
constexpr int threads = 256;

}  // namespace kv_rot

/*
Supports head size 32, 64, 128, 256
*/

template <typename T, int qRatio, int headSize, bool doRotary, int paddedHeadSize>
__global__ void kv_rotary_pos_kernel(T* kv_cache,
                                     T* q,
                                     T* k,
                                     T* v,
                                     const T* inv_freq,
                                     const int32_t rotary_dim,
                                     const float theta_base,
                                     const BatchWrapperCPP batch_desc,
                                     const int qkv_stride,
                                     const int kv_cache_stride,
                                     const int v_offset,
                                     const int inv_freq_stride)
{
    // Derived constexpr
    constexpr int vector_T = kv_rot::granularity / sizeof(T);
    constexpr int real_threads_per_head = headSize / vector_T;
    constexpr int threads_per_head = paddedHeadSize / vector_T;

    constexpr int tokens_per_block = kv_rot::threads / threads_per_head;

    // CG helpers
    cg::thread_block tb = cg::this_thread_block();
    cg::thread_block_tile<hw_warp_size> warp = cg::tiled_partition<hw_warp_size>(tb);
    cg::thread_block_tile<threads_per_head> head_group = cg::tiled_partition<threads_per_head>(tb);

    // Parallelize on the head dimension for X blocks
    const int head_idx = blockIdx.x;

    const int block_seq_idx = threadIdx.x / threads_per_head;
    const int base_neuron_idx = head_group.thread_rank() * vector_T;
    const int half_rotary_size = rotary_dim / 2;
    const int half_dim_lanes = half_rotary_size / vector_T;
    const int half_idx = base_neuron_idx % half_rotary_size;

    // Multiple tokens processed by the same threadblock
    const int token_idx = blockIdx.y * tokens_per_block + block_seq_idx;
    const bool valid_token = token_idx < batch_desc.batch_metadata->n_tokens;

    const bool valid_thread = valid_token && (head_group.thread_rank() < real_threads_per_head);
    const bool load_inv_freq = (inv_freq != nullptr) && valid_thread;

    // If we have GQA, then only one of the Q heads needs to do rotary + copy
    // for each of the heads in the group.
    bool need_kv = head_idx % qRatio == 0;
    // Make sure the following code is warp uniform
    need_kv = warp.shfl(need_kv, 0);

    const int kv_head_idx = head_idx / qRatio;

    // Ensure we don't access invalid portions of the seq_metadata
    const int32_t seq_id = (valid_thread) ? batch_desc.tokens_to_seq[token_idx] : 0;
    const InflightSeqDescriptor seq_desc = batch_desc.seq_metadata[seq_id];
    // This will give an invalid index if valid_thread is false, but should never affect memory.
    const int32_t global_token_idx = seq_desc.seen_tokens + (token_idx - seq_desc.start_idx);

    T* q_row = q + token_idx * qkv_stride + head_idx * headSize;
    T q_reg[vector_T];

    if (need_kv) {
        // The following logic assumes a linearly blocked KV cache. This means that no sparsity has
        // been introduced into cache history.
        const KVCacheDescriptor kv_desc = batch_desc.kv_desc;
        const int32_t seq_kv_block_idx = global_token_idx / kv_desc.block_size;
        const int32_t mapped_kv_block_idx =
            (valid_thread) ? kv_desc.block_lists[seq_id][seq_kv_block_idx] : 0;

        const int32_t kv_block_offset = global_token_idx % kv_desc.block_size;
        const int32_t kv_offset =
            (mapped_kv_block_idx * kv_desc.block_size + kv_block_offset) * kv_cache_stride +
            kv_head_idx * headSize;

        // Load indices from QKV output
        T* k_row = k + token_idx * qkv_stride + kv_head_idx * headSize;
        T* v_row = v + token_idx * qkv_stride + kv_head_idx * headSize;

        T k_reg[vector_T], v_reg[vector_T], inv_freq_reg[vector_T];

        mem_access::load_global<kv_rot::granularity>(q_reg, q_row + base_neuron_idx, valid_thread);
        mem_access::load_global<kv_rot::granularity>(k_reg, k_row + base_neuron_idx, valid_thread);
        mem_access::load_global<kv_rot::granularity>(v_reg, v_row + base_neuron_idx, valid_thread);
        mem_access::load_global<kv_rot::granularity>(
            inv_freq_reg, inv_freq + half_idx, load_inv_freq);
        if constexpr (doRotary) {
#pragma unroll
            for (int i = 0; i < vector_T; i++) {
                const int head_neuron_idx = base_neuron_idx + i;

                float inv_freq_flt;
                if (inv_freq != nullptr) {
                    inv_freq_flt = conversion::to<float>(inv_freq_reg[i]) * (float)global_token_idx;
                } else {
                    inv_freq_flt =
                        (float)((head_neuron_idx % half_rotary_size) * 2) / (float)rotary_dim;
                    // Conversion to T and back means that both branches of this if statement
                    // will produce the same results if using the same algo for producing the
                    // freqs.
                    T trunc_freq = conversion::to<T>(1.0 / powf(theta_base, inv_freq_flt));
                    inv_freq_flt = conversion::to<float>(trunc_freq) * (float)global_token_idx;
                }

                float rotary_sign = (head_neuron_idx >= half_rotary_size) ? -1.0f : 1.0f;
                float q_f = conversion::to<float>(q_reg[i]);
                float k_f = conversion::to<float>(k_reg[i]);
                float q_rot = q_f * rotary_sign;
                float k_rot = k_f * rotary_sign;

                const int target_lane = (head_neuron_idx < half_rotary_size)
                                            ? head_group.thread_rank() + half_dim_lanes
                                            : head_group.thread_rank() - half_dim_lanes;

                const float q_rot_temp = head_group.shfl(q_rot, target_lane);
                const float k_rot_temp = head_group.shfl(k_rot, target_lane);

                if (base_neuron_idx < rotary_dim) {
                    q_reg[i] = conversion::to<T>(q_f * cosf(inv_freq_flt) +
                                                 q_rot_temp * sinf(inv_freq_flt));
                    k_reg[i] = conversion::to<T>(k_f * cosf(inv_freq_flt) +
                                                 k_rot_temp * sinf(inv_freq_flt));
                }
            }
        }

        if (valid_thread) {
            mem_access::store_global<kv_rot::granularity>(kv_cache + kv_offset + base_neuron_idx,
                                                          k_reg);
            mem_access::store_global<kv_rot::granularity>(
                kv_cache + kv_offset + base_neuron_idx + v_offset, v_reg);
        }
    } else {
        T inv_freq_reg[vector_T];

        mem_access::load_global<kv_rot::granularity>(q_reg, q_row + base_neuron_idx, valid_thread);
        mem_access::load_global<kv_rot::granularity>(
            inv_freq_reg, inv_freq + half_idx, load_inv_freq);

        if constexpr (doRotary) {
#pragma unroll
            for (int i = 0; i < vector_T; i++) {
                const int head_neuron_idx = base_neuron_idx + i;

                float inv_freq_flt;
                if (inv_freq != nullptr) {
                    inv_freq_flt = conversion::to<float>(inv_freq_reg[i]) * (float)global_token_idx;
                } else {
                    inv_freq_flt =
                        (float)((head_neuron_idx % half_rotary_size) * 2) / (float)rotary_dim;
                    inv_freq_flt = 1.0 / powf(theta_base, inv_freq_flt) * (float)global_token_idx;
                }

                float rotary_sign = (head_neuron_idx >= half_rotary_size) ? -1.0f : 1.0f;
                float q_f = conversion::to<float>(q_reg[i]);
                float q_rot = q_f * rotary_sign;

                const int target_lane = (head_neuron_idx < half_rotary_size)
                                            ? head_group.thread_rank() + half_dim_lanes
                                            : head_group.thread_rank() - half_dim_lanes;

                const float q_rot_temp = head_group.shfl(q_rot, target_lane);
                if (base_neuron_idx < rotary_dim)
                    q_reg[i] = conversion::to<T>(q_f * cosf(inv_freq_flt) +
                                                 q_rot_temp * sinf(inv_freq_flt));
            }
        }
    }

    if (valid_thread && doRotary) {
        mem_access::store_global<kv_rot::granularity>(q_row + base_neuron_idx, q_reg);
    }
}

#define DISPATCH_KV_ROTARY_IMPL(Q_RATIO, HEAD_SIZE, PADDED_HEAD_SIZE)       \
    if (q_ratio == Q_RATIO && head_size == HEAD_SIZE)                       \
        kv_rotary_pos_kernel<T, Q_RATIO, HEAD_SIZE, true, PADDED_HEAD_SIZE> \
            <<<grid, block, 0, stream>>>(kv_cache,                          \
                                         q,                                 \
                                         k,                                 \
                                         v,                                 \
                                         inv_freq,                          \
                                         rotary_dim,                        \
                                         theta_base,                        \
                                         batch_desc,                        \
                                         qkv_stride,                        \
                                         kv_cache_stride,                   \
                                         v_offset,                          \
                                         inv_freq_stride);

#define LAUNCH_KV_ROTARY_FOR_Q_RATIO_HEAD_SIZE(Q_RATIO, HEAD_SIZE) \
    if (padded_head_size == 64) {                                  \
        DISPATCH_KV_ROTARY_IMPL(Q_RATIO, HEAD_SIZE, 64);           \
    } else if (padded_head_size == 128) {                          \
        DISPATCH_KV_ROTARY_IMPL(Q_RATIO, HEAD_SIZE, 128);          \
    } else {                                                       \
        assert(false);                                             \
    }

#define LAUNCH_KV_ROTARY_FOR_Q_RATIO(Q_RATIO)                 \
    if (head_size == 64) {                                    \
        LAUNCH_KV_ROTARY_FOR_Q_RATIO_HEAD_SIZE(Q_RATIO, 64);  \
    } else if (head_size == 80) {                             \
        LAUNCH_KV_ROTARY_FOR_Q_RATIO_HEAD_SIZE(Q_RATIO, 80);  \
    } else if (head_size == 96) {                             \
        LAUNCH_KV_ROTARY_FOR_Q_RATIO_HEAD_SIZE(Q_RATIO, 96);  \
    } else if (head_size == 128) {                            \
        LAUNCH_KV_ROTARY_FOR_Q_RATIO_HEAD_SIZE(Q_RATIO, 128); \
    } else {                                                  \
        assert(false);                                        \
    }

template <typename T>
void launch_kv_rotary_kernel(T* kv_cache,
                             T* q,
                             T* k,
                             T* v,
                             T* inv_freq,
                             const int32_t rotary_dim,
                             const float theta_base,
                             const BatchWrapperCPP batch_desc,
                             const int qkv_stride,
                             const int kv_cache_stride,
                             const int v_offset,
                             const int inv_freq_stride,
                             const int q_ratio,
                             const int head_size,
                             const int n_tokens,
                             const int n_q_heads,
                             hipStream_t stream)
{
    constexpr int vector_T = kv_rot::granularity / sizeof(T);

    const int padded_head_size = next_pow2(head_size);
    const int threads_per_head = padded_head_size / vector_T;

    const int tokens_per_block = kv_rot::threads / threads_per_head;

    const dim3 block(kv_rot::threads);
    const int token_blocks = (n_tokens + tokens_per_block - 1) / tokens_per_block;
    const dim3 grid(n_q_heads, token_blocks);

    LAUNCH_KV_ROTARY_FOR_Q_RATIO(1)
    LAUNCH_KV_ROTARY_FOR_Q_RATIO(2)
    LAUNCH_KV_ROTARY_FOR_Q_RATIO(4)
    LAUNCH_KV_ROTARY_FOR_Q_RATIO(5)
    LAUNCH_KV_ROTARY_FOR_Q_RATIO(6)
    LAUNCH_KV_ROTARY_FOR_Q_RATIO(7)
    LAUNCH_KV_ROTARY_FOR_Q_RATIO(8)
    LAUNCH_KV_ROTARY_FOR_Q_RATIO(16)
    LAUNCH_KV_ROTARY_FOR_Q_RATIO(29)
    LAUNCH_KV_ROTARY_FOR_Q_RATIO(35)
    LAUNCH_KV_ROTARY_FOR_Q_RATIO(36)
    LAUNCH_KV_ROTARY_FOR_Q_RATIO(71)
}

#define INSTANTIATE_KV_ROTARY_KERNEL(TYPE)                                        \
    template void launch_kv_rotary_kernel<TYPE>(TYPE * kv_cache,                  \
                                                TYPE * q,                         \
                                                TYPE * k,                         \
                                                TYPE * v,                         \
                                                TYPE * inv_freq,                  \
                                                const int32_t rotary_dim,         \
                                                const float theta_base,           \
                                                const BatchWrapperCPP batch_desc, \
                                                const int qkv_stride,             \
                                                const int kv_cache_stride,        \
                                                const int v_offset,               \
                                                const int inv_freq_stride,        \
                                                const int q_ratio,                \
                                                const int head_size,              \
                                                const int n_tokens,               \
                                                const int n_q_heads,              \
                                                hipStream_t stream);

INSTANTIATE_KV_ROTARY_KERNEL(__half)

#ifdef BF16_AVAILABLE
INSTANTIATE_KV_ROTARY_KERNEL(__hip_bfloat16)
#endif

#define DISPATCH_KV_COPY_IMPL(Q_RATIO, HEAD_SIZE, PADDED_HEAD_SIZE)          \
    if (q_ratio == Q_RATIO && head_size == HEAD_SIZE)                        \
        kv_rotary_pos_kernel<T, Q_RATIO, HEAD_SIZE, false, PADDED_HEAD_SIZE> \
            <<<grid, block, 0, stream>>>(kv_cache,                           \
                                         q,                                  \
                                         k,                                  \
                                         v,                                  \
                                         nullptr,                            \
                                         -1,                                 \
                                         0.f,                                \
                                         batch_desc,                         \
                                         qkv_stride,                         \
                                         kv_cache_stride,                    \
                                         v_offset,                           \
                                         0);

#define LAUNCH_KV_COPY_FOR_Q_RATIO_HEAD_SIZE(Q_RATIO, HEAD_SIZE) \
    if (padded_head_size == 64) {                                \
        DISPATCH_KV_COPY_IMPL(Q_RATIO, HEAD_SIZE, 64);           \
    } else if (padded_head_size == 128) {                        \
        DISPATCH_KV_COPY_IMPL(Q_RATIO, HEAD_SIZE, 128);          \
    } else {                                                     \
        assert(false);                                           \
    }

#define LAUNCH_KV_COPY_FOR_Q_RATIO(Q_RATIO)                 \
    if (head_size == 64) {                                  \
        LAUNCH_KV_COPY_FOR_Q_RATIO_HEAD_SIZE(Q_RATIO, 64);  \
    } else if (head_size == 80) {                           \
        LAUNCH_KV_COPY_FOR_Q_RATIO_HEAD_SIZE(Q_RATIO, 80);  \
    } else if (head_size == 96) {                           \
        LAUNCH_KV_COPY_FOR_Q_RATIO_HEAD_SIZE(Q_RATIO, 96);  \
    } else if (head_size == 128) {                          \
        LAUNCH_KV_COPY_FOR_Q_RATIO_HEAD_SIZE(Q_RATIO, 128); \
    } else {                                                \
        assert(false);                                      \
    }

template <typename T>
void launch_kv_copy_kernel(T* kv_cache,
                           T* q,
                           T* k,
                           T* v,
                           const BatchWrapperCPP batch_desc,
                           const int qkv_stride,
                           const int kv_cache_stride,
                           const int v_offset,
                           const int q_ratio,
                           const int head_size,
                           const int n_tokens,
                           const int n_q_heads,
                           hipStream_t stream)
{
    constexpr int vector_T = kv_rot::granularity / sizeof(T);
    const int padded_head_size = next_pow2(head_size);
    const int threads_per_head = padded_head_size / vector_T;
    const int tokens_per_block = kv_rot::threads / threads_per_head;

    const dim3 block(kv_rot::threads);
    const int token_blocks = (n_tokens + tokens_per_block - 1) / tokens_per_block;
    const dim3 grid(n_q_heads, token_blocks);

    LAUNCH_KV_COPY_FOR_Q_RATIO(1)
    LAUNCH_KV_COPY_FOR_Q_RATIO(2)
    LAUNCH_KV_COPY_FOR_Q_RATIO(4)
    LAUNCH_KV_COPY_FOR_Q_RATIO(5)
    LAUNCH_KV_COPY_FOR_Q_RATIO(8)
}

#define INSTANTIATE_KV_COPY_KERNEL(TYPE)                                        \
    template void launch_kv_copy_kernel<TYPE>(TYPE * kv_cache,                  \
                                              TYPE * q,                         \
                                              TYPE * k,                         \
                                              TYPE * v,                         \
                                              const BatchWrapperCPP batch_desc, \
                                              const int qkv_stride,             \
                                              const int kv_cache_stride,        \
                                              const int v_offset,               \
                                              const int q_ratio,                \
                                              const int head_size,              \
                                              const int n_tokens,               \
                                              const int n_q_heads,              \
                                              hipStream_t stream);

INSTANTIATE_KV_COPY_KERNEL(__half)

#ifdef BF16_AVAILABLE
INSTANTIATE_KV_COPY_KERNEL(__hip_bfloat16)
#endif
